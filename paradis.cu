#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

#define MAXIMUM_DIGITS 1
#define ARRAY_SIZE 10 
#define NUM_OF_PROCESSORS 4
#define NUM_OF_BUCKETS 10

__device__ int getBucket(int d_num, int d_level, int d_maxLevel=MAXIMUM_DIGITS)
{
    int i, powerOfTen=1;
    for(i=0; i<d_maxLevel-d_level-1; i++)
    {
        powerOfTen *= 10;
    }
    return (d_num/powerOfTen)%10;
}

__global__ void buildLocalHistogram(int *d_localHistogram, int *d_arr, int d_size, int d_level, int d_numOfBuckets, int d_numOfProcessors)
{
    int id =  blockIdx.x*blockDim.x+threadIdx.x;
    int i = id;
    while(i<d_size)
    {
        *(d_localHistogram + id*d_numOfBuckets + getBucket(d_arr[i], d_level)) += 1;
        i += d_numOfProcessors;
    }
}

__global__ void buildHistogram(int *d_histogram,int *d_localHistogram, int d_numOfBuckets, int d_numOfProcessors)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    for(int i=0;i<d_numOfProcessors;i++)
    {
        d_histogram[id] += *(d_localHistogram + i*d_numOfBuckets + id);
    }    
}

void paradis(int *h_arr, int size, int level, int numOfBuckets, int numOfProcessors)
{
    int *d_arr;
    //Allocate memory for array on device
    hipMalloc((void **)&d_arr, size*sizeof(int));
    //Copy array from host to device
    hipMemcpy((void *)d_arr, (void *)h_arr, size*sizeof(int), hipMemcpyHostToDevice);
    
    //
    // int *h_histogram, *h_localHistogram;   
    // h_histogram = (int *)malloc(sizeof(int)*numOfBuckets);
    // h_localHistogram = (int *)malloc(sizeof(int)*numOfBuckets*numOfProcessors);
    //

    int *d_histogram,  *d_localHistogram;
    hipMalloc((void **)&d_histogram, numOfBuckets*sizeof(int));
    hipMalloc((void **)&d_localHistogram, numOfBuckets*numOfProcessors*sizeof(int));

    hipMemset(d_histogram, 0, numOfBuckets*sizeof(int));
    hipMemset(d_localHistogram, 0, numOfBuckets*numOfProcessors*sizeof(int));

    buildLocalHistogram<<<1, numOfProcessors>>>(d_localHistogram, d_arr, size, level, numOfBuckets, numOfProcessors);
    buildHistogram<<<1, numOfBuckets>>>(d_histogram, d_localHistogram, numOfBuckets, numOfProcessors);

    hipMemcpy(h_histogram, d_histogram, numOfBuckets*sizeof(int), hipMemcpyDeviceToHost);    
    hipMemcpy(h_localHistogram, d_localHistogram, numOfBuckets*numOfProcessors*sizeof(int), hipMemcpyDeviceToHost);    

    // for(int i=0;i<numOfBuckets;i++)
    // {
    //     printf("%d: %d\n", i, h_histogram[i]);
    // }

    // for(int i=0;i<numOfProcessors;i++)
    // {
    //     for(int j=0;j<numOfBuckets;j++)
    //     {
    //         printf("%d\t", *(h_localHistogram + i*numOfBuckets + j));    
    //     }    
    //     printf("\n");
    // }        

    //Copy sorted array from device to host
    hipMemcpy((void *)h_arr, (void *)d_arr, ARRAY_SIZE*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_arr);
}

int main()
{
    int *h_arr;
    int i;

    //Allocate memory for array on host                
    h_arr = (int *)malloc(ARRAY_SIZE*sizeof(int));

    //Initialize array elements with random values    
    for(i=0; i<ARRAY_SIZE; i++)
    {
        h_arr[i] = abs(rand()%((int)pow(10, MAXIMUM_DIGITS)));
        printf("%d ", h_arr[i]);    
    }

    //Call sort function
    paradis(h_arr, ARRAY_SIZE, 0, NUM_OF_BUCKETS, NUM_OF_PROCESSORS);

    //Print the sorted array
    printf("\nSorted Array :\n");    
    for(i=0; i<ARRAY_SIZE; i++)
    {
        printf("%d ", h_arr[i]);
    }
    printf("\n");
    //Free memory
    free(h_arr);

    return 0;
}
