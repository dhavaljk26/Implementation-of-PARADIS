#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

#define MAXIMUM_DIGITS 1
#define ARRAY_SIZE 10 
#define NUM_OF_PROCESSORS 4
#define NUM_OF_BUCKETS 10

__device__ int getBucket(int d_num, int d_level, int d_maxLevel=MAXIMUM_DIGITS)
{
    int i, powerOfTen=1;
    for(i=0; i<d_maxLevel-d_level-1; i++)
    {
        powerOfTen *= 10;
    }
    return (d_num/powerOfTen)%10;
}

__global__ void buildLocalHistogram(int *d_localHistogram, int *d_arr, int d_size, int d_level, int d_numOfBuckets, int d_numOfProcessors)
{
    int id =  blockIdx.x*blockDim.x+threadIdx.x;
    int i = id;
    while(i<d_size)
    {
        *(d_localHistogram + id*d_numOfBuckets + getBucket(d_arr[i], d_level)) += 1;
        i += d_numOfProcessors;
    }
}

__global__ void buildHistogram(int *d_histogram,int *d_localHistogram, int d_numOfBuckets, int d_numOfProcessors)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    for(int i=0;i<d_numOfProcessors;i++)
    {
        d_histogram[id] += *(d_localHistogram + i*d_numOfBuckets + id);
    }    
}

__global__ void prefixSum(int *d_out, int*d_out2, int *d_in, int n)
{
    // extern __shared__ float temp[]; 
    // int id = threadIdx.x;
    // int pout = 0, pin = 1;
    
    // temp[pout*n + id] = (id > 0) ? d_in[id-1] : 0;
    // __syncthreads();
    
    // for (int offset=1; offset<n; offset*=2)
    // {
    //     pout = 1 - pout; 
    //     pin = 1 - pout;
    //     if (id >= offset)
    //         temp[pout*n+id] += temp[pin*n+id - offset];
    //     else
    //         temp[pout*n+id] = temp[pin*n+id];
    //     __syncthreads();
    // }

    // d_out[id] = temp[pout*n+id];

    d_out[0] = 0;
    d_out2[0] = d_in[0];
    for(int i=1;i<n;i++)
    {
        d_out[i] = d_out[i-1] + d_in[i-1];
        d_out2[i] = d_out2[i-1] + d_in[i];
    }
    
}

void paradis(int *h_arr, int size, int level, int numOfBuckets, int numOfProcessors)
{
    int *d_arr;
    //Allocate memory for array on device
    hipMalloc((void **)&d_arr, size*sizeof(int));
    //Copy array from host to device
    hipMemcpy((void *)d_arr, (void *)h_arr, size*sizeof(int), hipMemcpyHostToDevice);
    
    //
    int *h_histogram, *h_localHistogram, *h_gh, *h_gt;   
    h_histogram = (int *)malloc(sizeof(int)*numOfBuckets);
    // h_localHistogram = (int *)malloc(sizeof(int)*numOfBuckets*numOfProcessors);
    h_gh = (int *)malloc(sizeof(int)*numOfBuckets);
    h_gt = (int *)malloc(sizeof(int)*numOfBuckets);

    int *d_histogram,  *d_localHistogram, *d_gh, *d_gt;
 
    hipMalloc((void **)&d_histogram, numOfBuckets*sizeof(int));
    hipMalloc((void **)&d_localHistogram, numOfBuckets*numOfProcessors*sizeof(int));
    hipMalloc((void **)&d_gh, numOfBuckets*sizeof(int));
    hipMalloc((void **)&d_gt, numOfBuckets*sizeof(int));
    
    hipMemset(d_histogram, 0, numOfBuckets*sizeof(int));
    hipMemset(d_localHistogram, 0, numOfBuckets*numOfProcessors*sizeof(int));

    buildLocalHistogram<<<1, numOfProcessors>>>(d_localHistogram, d_arr, size, level, numOfBuckets, numOfProcessors);
    buildHistogram<<<1, numOfBuckets>>>(d_histogram, d_localHistogram, numOfBuckets, numOfProcessors);

    prefixSum<<<1, 1>>>(d_gh, d_gt, d_histogram, numOfBuckets);    

    hipMemcpy(h_gh, d_gh, numOfBuckets*sizeof(int), hipMemcpyDeviceToHost);    
    hipMemcpy(h_gt, d_gt, numOfBuckets*sizeof(int), hipMemcpyDeviceToHost);    

    // cudaMemcpy(h_localHistogram, d_localHistogram, numOfBuckets*numOfProcessors*sizeof(int), cudaMemcpyDeviceToHost);    
    hipMemcpy(h_histogram, d_histogram, numOfBuckets*sizeof(int), hipMemcpyDeviceToHost); 

    for(int i=0;i<numOfBuckets;i++)
    {
        printf("\n%d: hist=%d c1=%d c2=%d", i, h_histogram[i], h_gh[i], h_gt[i]);
    }
    
    // for(int i=0;i<numOfProcessors;i++)
    // {
    //     for(int j=0;j<numOfBuckets;j++)
    //     {
    //         printf("%d\t", *(h_localHistogram + i*numOfBuckets + j));    
    //     }    
    //     printf("\n");
    // }
    

    //Copy sorted array from device to host
    hipMemcpy((void *)h_arr, (void *)d_arr, ARRAY_SIZE*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_arr);
}

int main()
{
    int *h_arr;
    int i;

    //Allocate memory for array on host                
    h_arr = (int *)malloc(ARRAY_SIZE*sizeof(int));

    //Initialize array elements with random values    
    for(i=0; i<ARRAY_SIZE; i++)
    {
        h_arr[i] = abs(rand()%((int)pow(10, MAXIMUM_DIGITS)));
        printf("%d ", h_arr[i]);    
    }

    //Call sort function
    paradis(h_arr, ARRAY_SIZE, 0, NUM_OF_BUCKETS, NUM_OF_PROCESSORS);

    //Print the sorted array
    printf("\nSorted Array :\n");    
    for(i=0; i<ARRAY_SIZE; i++)
    {
        printf("%d ", h_arr[i]);
    }
    printf("\n");
    //Free memory
    free(h_arr);

    return 0;
}
